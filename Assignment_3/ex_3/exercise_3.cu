#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <inttypes.h>
#include <ctime>


typedef struct {
	float3 position = float3{ 1, 1, 1 };
	float3 velocity = float3{ 1, 1, 1 };
} Particle;

static uint64_t time_ns(void)
{
	struct timespec ts;

	if (timespec_get(&ts, TIME_UTC) != TIME_UTC)
	{
		fputs("timespec_get failed!", stderr);
		return 0;
	}
	return (uint64_t)1e9 * ts.tv_sec + ts.tv_nsec;
}

void setup_data(Particle* list, size_t size, unsigned int seed)
{
	srand(seed);
	for (size_t i = 0; i < size; i++)
	{
		list[i].position = float3{
			rand() / (float)RAND_MAX,
			rand() / (float)RAND_MAX,
			rand() / (float)RAND_MAX
		};
		list[i].velocity = float3{
			rand() / (float)RAND_MAX,
			rand() / (float)RAND_MAX,
			rand() / (float)RAND_MAX
		};
	}
}

__device__ float3 update_position(float3 position, float3 velocity, float dt)
{
	position.x += velocity.x * dt;
	position.y += velocity.y * dt;
	position.z += velocity.z * dt;
	return position;
}

__device__ float3 update_velocity(float3 velocity, float dt, size_t iteration)
{
	float acc = (iteration % 100 == 0) ? 0.1f : 0.0f;
	velocity.x += acc;
	velocity.y += acc;
	velocity.z += acc;
	return velocity;
}

__global__ void simpleKernel(Particle* d_particles, size_t array_size, time_t seed, hiprandState* states, size_t iteration, size_t offset)
{
	const int i = offset + blockIdx.x * blockDim.x + threadIdx.x;
	float dt = 1.0f;

	if (i >= array_size) return;

	d_particles[i].velocity = update_velocity(d_particles[i].velocity, dt, iteration);
	d_particles[i].position = update_position(d_particles[i].position, d_particles[i].velocity, dt);
}

void execute_on_GPU(Particle* particles, time_t seed, size_t number_of_iterations, size_t number_of_particles, size_t block_size, size_t number_of_streams)
{
	printf("Computing particles on the GPU...\n");
	uint64_t start = time_ns();

	Particle* d_particles = 0;
	hipMalloc(&d_particles, number_of_particles * sizeof(Particle));

	hiprandState* d_randoms;
	hipMalloc((void**)&d_randoms, number_of_particles * sizeof(hiprandState));

	const int stream_size = number_of_particles / number_of_streams;
	const int stream_bytes = stream_size * sizeof(Particle);

	hipStream_t* streams = (hipStream_t*)malloc(number_of_streams * sizeof(hipStream_t));
	for (size_t i = 0; i < number_of_streams; i++)
		hipStreamCreate(&streams[i]);

	for (size_t j = 0; j < number_of_iterations; j++)
	{
		for (size_t i = 0; i < number_of_streams; i++)
		{
			const int offset = i * stream_size;

			hipMemcpyAsync(&d_particles[offset], &particles[offset], stream_bytes, hipMemcpyHostToDevice, streams[i]);

			simpleKernel << <(number_of_particles + block_size - 1) / block_size, block_size, 0, streams[i] >> > (d_particles, number_of_particles, seed, d_randoms, j, offset);

			hipMemcpyAsync(&particles[offset], &d_particles[offset], stream_bytes, hipMemcpyDeviceToHost, streams[i]);
		}
	}

	hipDeviceSynchronize();

	for (size_t i = 0; i < number_of_streams; i++)
		hipStreamDestroy(streams[i]);

	uint64_t end = time_ns() - start;
	printf("Done! Time elapsed: (ns) = %llu\t (s): %f\n\n", end, (float)end / 1e9);

	hipFree(d_particles);
	hipFree(d_randoms);
}

int main(int argc, char* argv[])
{
	if (argc != 5)
	{
		printf("Not enough arguments <number of particles> <number of iterations> <block size (threads per block)>...\n");
		return -1;
	}
	size_t num_particles = strtoull(argv[1], NULL, 10);
	size_t num_iterations = strtoull(argv[2], NULL, 10);
	size_t block_size = strtoull(argv[3], NULL, 10);
	size_t number_of_streams = strtoull(argv[4], NULL, 10);

	printf("Execution in progress with setup:\n\tnumber of particles: %d, number of iterations: %d, block size (threads per block): %d...\n", num_particles, num_iterations, block_size);

	Particle* h_data = 0;

	hipHostMalloc(&h_data, num_particles * sizeof(Particle), hipHostMallocDefault);

	time_t seed = time(NULL);
	setup_data(h_data, num_particles, seed);

	execute_on_GPU(h_data, seed, num_iterations, num_particles, block_size, number_of_streams);

	hipHostFree(h_data);

	return 0;
}