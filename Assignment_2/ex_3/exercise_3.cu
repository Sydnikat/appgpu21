#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <inttypes.h>
#include <ctime>

typedef struct {
	float3 position = float3{ 1, 1, 1 };
	float3 velocity = float3{ 1, 1, 1 };
} Particle;

static uint64_t time_ns(void)
{
	struct timespec ts;

	if (timespec_get(&ts, TIME_UTC) != TIME_UTC)
	{
		fputs("timespec_get failed!", stderr);
		return 0;
	}
	return (uint64_t)1e9 * ts.tv_sec + ts.tv_nsec;
}

void setup_data(Particle* list, size_t size, unsigned int seed)
{
	srand(seed);
	for (size_t i = 0; i < size; i++)
	{
		list[i].position = float3{
			rand() / (float)RAND_MAX,
			rand() / (float)RAND_MAX,
			rand() / (float)RAND_MAX
		};
		list[i].velocity = float3{
			rand() / (float)RAND_MAX,
			rand() / (float)RAND_MAX,
			rand() / (float)RAND_MAX
		};
	}
}

__host__ __device__ float3 update_position(float3 position, float3 velocity, float dt)
{
	position.x += velocity.x * dt;
	position.y += velocity.y * dt;
	position.z += velocity.z * dt;
	return position;
}

__host__ __device__ float3 update_velocity(float3 velocity, float dt, size_t iteration)
{
	float acc = (iteration % 100 == 0) ? 0.1f : 0.0f;
	velocity.x += acc;
	velocity.y += acc;
	velocity.z += acc;
	return velocity;
}

void execute_on_CPU(Particle* particles, time_t seed, size_t number_of_iterations, size_t number_of_particles)
{

	printf("Computing particles on the CPU...\n");

	uint64_t start = time_ns();
	float dt = 1.0f;

	for (size_t j = 0; j < number_of_iterations; j++)
	{
		for (size_t i = 0; i < number_of_particles; i++)
		{
			particles[i].velocity = update_velocity(particles[i].velocity, dt, j);
			particles[i].position = update_position(particles[i].position, particles[i].velocity, dt);
		}
	}

	uint64_t end = time_ns() - start;
	printf("Done! Time elapsed: (ns) = %lu\t (s): %f\n\n", end, (float)end / 1e9);
}



__global__ void particleKernel(Particle* d_particles, size_t array_size, time_t seed, hiprandState* states, size_t number_of_iterations)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	float dt = 1.0f;

	if (i >= array_size) return;

	for (size_t j = 0; j < number_of_iterations; j++)
	{
		d_particles[i].velocity = update_velocity(d_particles[i].velocity, dt, j);
		d_particles[i].position = update_position(d_particles[i].position, d_particles[i].velocity, dt);
	}
}

void execute_on_GPU(Particle* particles, time_t seed, size_t number_of_iterations, size_t number_of_particles, size_t block_size)
{
	printf("Computing particles on the GPU...\n");
	uint64_t start = time_ns();

	Particle* d_particles = 0;
	hipMalloc(&d_particles, number_of_particles * sizeof(Particle));

	hiprandState* d_randoms;
	hipMalloc((void**)&d_randoms, number_of_particles * sizeof(hiprandState));


	hipMemcpy(d_particles, particles, number_of_particles * sizeof(Particle), hipMemcpyHostToDevice);
	 
	particleKernel << <(number_of_particles + block_size - 1) / block_size, block_size >> >(d_particles, number_of_particles, seed, d_randoms, number_of_iterations);

	hipDeviceSynchronize();

	hipMemcpy(particles, d_particles, number_of_particles * sizeof(Particle), hipMemcpyDeviceToHost);

	uint64_t end = time_ns() - start;
	printf("Done! Time elapsed: (ns) = %d\t (s): %f\n\n", end, (float)end / 1e9);

	hipFree(d_particles);
	hipFree(d_randoms);
}

bool equals(const float3 a, const float3 b)
{
	const float margin = 1e-6;
	return fabs(a.x - b.x) < margin && fabs(a.y - b.y) < margin && fabs(a.z - b.z) < margin;
}

int main(int argc, char* argv[])
{
	if (argc != 4)
	{
		printf("Not enough arguments <number of particles> <number of iterations> <block size (threads per block)>...\n");
		return -1;
	}
	size_t num_particles = strtoull(argv[1], NULL, 10);
	size_t num_iterations = strtoull(argv[2], NULL, 10);
	size_t block_size = strtoull(argv[3], NULL, 10);

	printf("Execution in progress with setup:\n\tnumber of particles: %d, number of iterations: %d, block size (threads per block): %d...\n", num_particles, num_iterations, block_size);

	Particle* cpu_data = (Particle*)malloc(num_particles * sizeof(Particle));
	Particle* gpu_data = (Particle*)malloc(num_particles * sizeof(Particle));

	time_t seed = time(NULL);
	setup_data(cpu_data, num_particles, seed);
	setup_data(gpu_data, num_particles, seed);

	execute_on_CPU(cpu_data, seed, num_iterations, num_particles);

	execute_on_GPU(gpu_data, seed, num_iterations, num_particles, block_size);

	printf("Comparing the output for each implementation for %d particles...", num_particles);
	bool same = true;
	for (size_t i = 0; i < num_particles; i++)
	{
		float3 c_v = cpu_data[i].velocity;
		float3 c_p = cpu_data[i].position;
		float3 g_v = gpu_data[i].velocity;
		float3 g_p = gpu_data[i].position;
		if (!equals(c_p, g_p) || !equals(c_v, g_v))
		{
			same = false;
			break;
		}
	}

	printf("\t%s!\n", same ? "Correct" : "Failed!");

	free(cpu_data);
	free(gpu_data);

	return 0;
}